#include "hip/hip_runtime.h"
#include "stdio.h"
#include "TvDenoise.h"
#include "D:\Matlab\extern\include\mex.h"

#define BLOCK_SIZE 16

// c��c++�еľ����������ȴ洢
// M.width ��ƫ��
// M.elements �����׵�ַ
// M(row, col) = *(M.elements + col * M.height + row)
typedef struct
{
    int width; // ����
    int height; // ����
    int band; // �׶���
    int size; // �г�����
    float* elements; // ��ʼָ��
} Matrix;

// ����������к������ɾ���ṹ��
Matrix createMat(const int height, const int width, const int band)
{
    Matrix mat;
    mat.height = height;
    mat.width = width;
    mat.band = band;
    mat.size = height * width;
    return mat;
}

__constant__ float tau = 0.25;

__global__ void TvDenoiseGPUSetZero(Matrix A, float *dhvt, float *dhtp, float *dvtp)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

    if(x_id < A.width && y_id < A.height)
    {
        int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��
        dhvt[iIdx] = 0;
        dhtp[iIdx] = 0;
        dvtp[iIdx] = 0;
    }
}

__global__ void TvDenoiseGPUStepOne(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

    if(x_id < A.width && y_id < A.height)
    {
        int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

        // �� dhvt = opQt(pn)
        dhvt[iIdx] = dhtp[((x_id + A.width - 1) % A.width) * A.height + y_id] - dhtp[iIdx]
                     + dvtp[x_id * A.height + (y_id + A.height - 1) % A.height] - dvtp[iIdx];

        // �� dhvt = -opQt(pn) - g ./ lambda
        dhvt[iIdx] = - dhvt[iIdx] - A.elements[iIdx] / lambda;
    }
}

__global__ void TvDenoiseGPUStepTwo(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

    if(x_id < A.width && y_id < A.height)
    {
        int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

        // �� S = opQ(dhvt) = opQ(-opQt(pn) - g ./ lambda)
        float dh = dhvt[((x_id + 1) % A.width) * A.height + y_id] - dhvt[iIdx];
        float dv = dhvt[x_id * A.height + (y_id + 1) % A.height] - dhvt[iIdx];

        // �� R = (1 + tau * modulo(S))
        float R = 1 + tau * sqrt(dh * dh + dv * dv);

        // �� pn = [dht dvt]
        dhtp[iIdx] = (dhtp[iIdx] + tau * dh) / R;
        dvtp[iIdx] = (dvtp[iIdx] + tau * dv) / R;
    }
}

__global__ void TvDenoiseGPUStepThree(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

    if(x_id < A.width && y_id < A.height)
    {
        int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

        // �� dhvt = opQt(pn)
        dhvt[iIdx] = dhtp[((x_id + A.width - 1) % A.width) * A.height + y_id] - dhtp[iIdx]
                      + dvtp[x_id * A.height + (y_id + A.height - 1) % A.height] - dvtp[iIdx];
        // ��������д��A������
        A.elements[iIdx] += lambda * dhvt[iIdx];
    }
}

void TvDenoiseGPU(float *A, float *C, int iRow, int iCol, int band, float lambda, int piter)
{
    Matrix d_A = createMat(iRow, iCol, band); // ָ���Դ�ľ���

    size_t oneBandSize = d_A.size * sizeof(float); // һ���׶�ռ���ڴ�ռ��С
    size_t allBandSize = d_A.band * oneBandSize; // �����׶�ռ���ڴ�ռ��С

    // �����м��������ͷָ��
    float *dhvt, *dhtp, *dvtp; // �м����

    // ����Pageable(����ҳ)�ڴ�
    hipMalloc((void**)&d_A.elements, oneBandSize);// ���Դ���Ϊd_A���ٿռ�
    hipMalloc((void**)&dhvt, oneBandSize);// ���Դ���Ϊdhvt���ٿռ�
    hipMalloc((void**)&dhtp, oneBandSize);// ���Դ���Ϊdhtp���ٿռ�
    hipMalloc((void**)&dvtp, oneBandSize);// ���Դ���Ϊdvtp���ٿռ�

    float *CHost, *AHost;
    // ����Pinned(Page-locked)�ڴ�
    hipHostAlloc((void**)&CHost, allBandSize, hipHostMallocDefault);
    hipHostAlloc((void**)&AHost, allBandSize, hipHostMallocDefault);

    // ��δ��ҳ���ڴ�A��������ҳ���ڴ�AHost
    memcpy(AHost, A, allBandSize);

    // �˺����߶�
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((d_A.width + BLOCK_SIZE -1) / dimBlock.x,
                 (d_A.height + BLOCK_SIZE -1) / dimBlock.y);

    // ������
    hipStream_t stream; //���Ľṹ��
    hipStreamCreate(&stream);

    for(int iBand = 0; iBand < d_A.band; iBand++)
    {
        int iBandOffset = iBand * d_A.size;

        hipMemcpyAsync(d_A.elements, AHost + iBandOffset, oneBandSize, hipMemcpyHostToDevice, stream);

        TvDenoiseGPUSetZero<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp); // ��ʼ��htvt, dhtp, dvtp = 0;

        for (int iter = 0; iter < piter; ++iter)
        {
            TvDenoiseGPUStepOne<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp, lambda); // �� dhvt = -opQt(pn) - g ./ lambda
            TvDenoiseGPUStepTwo<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp, lambda);
            hipStreamSynchronize(stream);
        }
        TvDenoiseGPUStepThree<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp, lambda);

        // ��d_A������Դ��ж��������ڴ���
        hipMemcpyAsync(CHost + iBandOffset, d_A.elements, oneBandSize, hipMemcpyDeviceToHost, stream);
    }

    // ǿ��CUDA����ʱ�ȴ��������в������
    hipStreamSynchronize(stream);

    // ����ҳ���ڴ�CHost������δ��ҳ���ڴ�C
    memcpy(C, CHost, allBandSize);

    // �ͷ�Pinned(Page-locked)�ڴ�
    hipHostFree(CHost);
    hipHostFree(AHost);

    // �ͷ��Դ�ռ�
    hipFree(d_A.elements);
    hipFree(dhvt);
    hipFree(dhtp);
    hipFree(dvtp);

    hipDeviceReset();
}
