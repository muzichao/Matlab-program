#include "hip/hip_runtime.h"
#include "mulMatrixs.h"
#include "D:\Matlab\extern\include\mex.h"

#define BLOCK_SIZE 32

// c��c++�еľ����������ȴ洢
// M.stride ��ƫ��
// �˴�����ƫ����Ϊ�˷���ȡ������ӿ�
// M.elements �����׵�ַ
// M(row, col) = *(M.elements + col * M.stride + row)
typedef struct 
			{
				int width;
				int height;
				int stride;
				size_t size;
				float* elements;
			} Matrix;

// ����������к������ɾ���ṹ��
Matrix createMat(const int height, const int width)
{
	Matrix mat;
	mat.height = height;
	mat.width = width;
	mat.stride = height;
	mat.size = height * width * sizeof(float);
	return mat;
}

// ��þ���mat��(row, col)����Ԫ��
// �����ж���Ϊ�˷�ֹԽ��
// ���磺�����С�����߳̿�������ʱ�����ٵ��߳��Ǵ��ھ���Ԫ������
__device__ float GetElement(const Matrix mat, int row, int col)
{
	if(row < mat.height && col < mat.width)
		return mat.elements[col * mat.stride + row];
	else
		return 0;
}

// ���þ���mat��(row, col)����Ԫ��
// �����ж���Ϊ�˷�ֹԽ��
// ���磺�����С�����߳̿�������ʱ�����ٵ��߳��Ǵ��ھ���Ԫ������
__device__ void SetElement(Matrix mat, int row, int col, float value)
{
	if(row < mat.height && col < mat.width)
		mat.elements[col * mat.stride + row] = value;
}

// ���mat��һ���Ӿ���matSub����СΪ��BLOCK_SIZE * BLOCK_SIZE 
// �Ӿ���������Ϊ��row�� col��
__device__ Matrix GetSubMatrix(Matrix mat, int row, int col)
{
	Matrix matSub;
	matSub.width = BLOCK_SIZE;
	matSub.height = BLOCK_SIZE;
	matSub.stride = mat.stride; // �Ӿ�����ԭ�������ƫ����ͬ
	matSub.elements = &mat.elements[mat.stride * BLOCK_SIZE * col + BLOCK_SIZE * row];
	return matSub;
}

// ������˵ĺ˺���
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// ���Ӧ���к���
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	// ÿһ���̼߳���C�е�һ��Ԫ��
	// ���������Cvalue
	float Cvalue = 0;

	// �ӿ����̵߳�����
	int row = threadIdx.y;
	int col = threadIdx.x;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	// һ��ѭ���� A�����ӿ��е�һ���ӿ� * B�����ӿ��е�һ���ӿ� ����ӣ�
	// �ӿ��ѭ��
	for (int m = 0; m <  ((A.width + BLOCK_SIZE -1) / BLOCK_SIZE); ++m) 
	{
		// �õ�A�е�һ���ӿ�Asub
		Matrix Asub = GetSubMatrix(A, blockRow, m);

		// �õ�B�е�һ���ӿ�Bsub
		Matrix Bsub = GetSubMatrix(B, m, blockCol);

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// ��Asub��Bsub�����������ڴ���
		// ÿһ���߳̿����ӿ��е�һ��Ԫ��
		// ��Ϊ���һ���ӿ�����һ���ӿ����δ��
		// ��˽����жϣ���ֹԽ��
		if((m * BLOCK_SIZE + col < A.width) &&
			(blockRow * BLOCK_SIZE + row < A.height))
		{
			As[row][col] = GetElement(Asub, row, col);
		}
		else
		{
			As[row][col] = 0;
		}

		if((blockCol * BLOCK_SIZE + col < B.width) &&
			(m * BLOCK_SIZE + row < B.height))
		{
			Bs[row][col] = GetElement(Bsub, row, col);
		}
		else
		{
			Bs[row][col] = 0;
		}

		// ���߳̿��е��߳̽���ͬ����ȷ���߳̿��е�ÿ���̶߳�ִ����
		// �˴�ͬ����Ϊ��ȷ���Ӿ����Ѿ������������ڴ���
		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int e = 0; e < BLOCK_SIZE; ++e)
		{
			Cvalue += As[row][e] * Bs[e][col];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		// ͬ����ɲŽ��¸�A�ӿ���B�ӿ鿽���Ĺ����ڴ�
		__syncthreads();
	}

	// C�ӿ�Csub�ļ��������
	// ÿ���߳�дһ��Ԫ��
	//SetElement(Csub, row, col, Cvalue);
	SetElement(C, blockRow * blockDim.y + row, blockCol * blockDim.x + col, Cvalue);
}

void mulMatrixs(float* A, float* B, float* C, int numRowsA, int numColsA, int numColsB)
{
    Matrix dev_A, dev_B, dev_C;
    dev_A = createMat(numRowsA, numColsA);
    dev_B = createMat(numColsA, numColsB);
    dev_C = createMat(numRowsA, numColsB);

    hipMalloc(&dev_A.elements, dev_A.size);
    hipMalloc(&dev_B.elements, dev_B.size);
    hipMalloc(&dev_C.elements, dev_C.size);
    
    hipMemcpy(dev_A.elements, A, dev_A.size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B.elements, B, dev_B.size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(int((numColsB + BLOCK_SIZE -1 )/BLOCK_SIZE),
                 int((numRowsA + BLOCK_SIZE -1 )/BLOCK_SIZE));
    MatMulKernel<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C);
    
    hipMemcpy(C, dev_C.elements, dev_C.size, hipMemcpyDeviceToHost);
    
    hipFree(dev_A.elements);
    hipFree(dev_B.elements);
    hipFree(dev_C.elements);   
}
