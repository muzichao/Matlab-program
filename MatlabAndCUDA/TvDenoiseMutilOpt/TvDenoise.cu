#include "hip/hip_runtime.h"
#include "TvDenoise.h"
#include "D:\Matlab\extern\include\mex.h"

#define BLOCK_SIZE 32

// c��c++�еľ����������ȴ洢
// M.width ��ƫ��
// M.elements �����׵�ַ
// M(row, col) = *(M.elements + col * M.height + row)
typedef struct
{
	int width; // ����
	int height; // ����
    int band; // �׶���
	int size; // �г�����
	float* elements; // ��ʼָ��
} Matrix;

// ����������к������ɾ���ṹ��
Matrix createMat(const int height, const int width, const int band)
{
	Matrix mat;
	mat.height = height;
	mat.width = width;
    mat.band = band;
	mat.size = height * width;
	return mat;
}

__constant__ float tau = 0.25;

__global__ void TvDenoiseGPUStepOne(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda, int piter, int iBand)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if(x_id < A.width && y_id < A.height)
	{
		int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

		// �� dhvt = opQt(pn)
        dhvt[iIdx] = dhtp[((x_id + A.width - 1) % A.width) * A.height + y_id] - dhtp[iIdx]
                     + dvtp[x_id * A.height + (y_id + A.height - 1) % A.height] - dvtp[iIdx];

		// �� dhvt = -opQt(pn) - g ./ lambda
		dhvt[iIdx] = - dhvt[iIdx] - A.elements[A.size * iBand + iIdx] / lambda;
	}
}

__global__ void TvDenoiseGPUStepTwo(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda, int piter)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if(x_id < A.width && y_id < A.height)
	{
		int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

		// �� S = opQ(dhvt) = opQ(-opQt(pn) - g ./ lambda)
        float dh = dhvt[((x_id + 1) % A.width) * A.height + y_id] - dhvt[iIdx];
        float dv = dhvt[x_id * A.height + (y_id + 1) % A.height] - dhvt[iIdx];

		// �� R = (1 + tau * modulo(S))
		float R = 1 + tau * sqrt(dh * dh + dv * dv);

		// �� pn = [dht dvt]
		dhtp[iIdx] = (dhtp[iIdx] + tau * dh) / R;
		dvtp[iIdx] = (dvtp[iIdx] + tau * dv) / R;
	}
}

__global__ void TvDenoiseGPUStepThree(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda, int piter, int iBand)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if(x_id < A.width && y_id < A.height)
	{
		int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

		// �� dhvt = opQt(pn)
        dhvt[iIdx] = dhtp[((x_id + A.width - 1) % A.width) * A.height + y_id] - dhtp[iIdx]
                      + dvtp[x_id * A.height + (y_id + A.height - 1) % A.height] - dvtp[iIdx];
		// ��������д��A������
		A.elements[A.size * iBand + iIdx] += lambda * dhvt[iIdx];
	}
}

void TvDenoiseGPU(float *A, float *C, int iRow, int iCol, int band, float lambda, int piter)
{
    Matrix d_A = createMat(iRow, iCol, band); // ָ���Դ�ľ���

    size_t oneBandSize = d_A.size * sizeof(float); // һ���׶�ռ���ڴ�ռ��С

    hipMalloc((void**)&d_A.elements, d_A.band * oneBandSize);// ���Դ���Ϊd_A���ٿռ�
    hipMemcpy(d_A.elements, A, oneBandSize * band, hipMemcpyHostToDevice);

    float *dhvt; // �м����
    float *dhtp; // �м����
    float *dvtp; // �м����
	hipMalloc((void**)&dhvt, oneBandSize);// ���Դ���Ϊdhvt���ٿռ�
    hipMalloc((void**)&dhtp, oneBandSize);// ���Դ���Ϊdhtp���ٿռ�
    hipMalloc((void**)&dvtp, oneBandSize);// ���Դ���Ϊdvtp���ٿռ�

    // �˺���
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((d_A.width + BLOCK_SIZE -1) / dimBlock.x,
                 (d_A.height + BLOCK_SIZE -1) / dimBlock.y);

    for(int iBand = 0; iBand < band; iBand++)
    {
        hipMemset(dhvt, 0, oneBandSize); //��ʼ��dhvt
        hipMemset(dhtp, 0, oneBandSize); //��ʼ��dhtp
        hipMemset(dvtp, 0, oneBandSize); //��ʼ��dvtp

        for (int iter = 0; iter < piter; ++iter)
        {
            TvDenoiseGPUStepOne<<<dimGrid, dimBlock>>>(d_A, dhvt, dhtp, dvtp, lambda, piter, iBand); // �� dhvt = -opQt(pn) - g ./ lambda
            TvDenoiseGPUStepTwo<<<dimGrid, dimBlock>>>(d_A, dhvt, dhtp, dvtp, lambda, piter);
        }
        TvDenoiseGPUStepThree<<<dimGrid, dimBlock>>>(d_A, dhvt, dhtp, dvtp, lambda, piter, iBand);
    }

    // ��d_A������Դ��ж��������ڴ���
    hipMemcpy(C, d_A.elements, oneBandSize * band, hipMemcpyDeviceToHost);

    // �ͷ��Դ�ռ�
    hipFree(d_A.elements);
	hipFree(dhvt);
	hipFree(dhtp);
	hipFree(dvtp);

    hipDeviceReset();
}
