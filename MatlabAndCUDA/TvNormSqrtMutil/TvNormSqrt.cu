#include "hip/hip_runtime.h"
#include "stdio.h"
#include "TvNormSqrt.h"
#include "D:\Matlab\extern\include\mex.h"

#define BLOCK_SIZE 16

// c和c++中的矩阵都是行优先存储
// M.width 行偏移
// M.elements 矩阵首地址
// M(row, col) = *(M.elements + col * M.height + row)
typedef struct
{
    int width; // 列数
    int height; // 行数
    int band; // 谱段数
    int size; // 行乘以列
    float* elements; // 起始指针
} Matrix;

// 根据输入的行和列生成矩阵结构体
Matrix createMat(const int height, const int width, const int band)
{
    Matrix mat;
    mat.height = height;
    mat.width = width;
    mat.band = band;
    mat.size = height * width;
    return mat;
}

__constant__ float tau = 0.25;

__global__ void TvDenoiseGPUSetZero(Matrix A, float *dhvt, float *dhtp, float *dvtp)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // 列坐标
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // 行坐标

    if(x_id < A.width && y_id < A.height)
    {
        int iIdx = x_id * A.height + y_id; // 该线程对应矩阵索引位置
        dhvt[iIdx] = 0;
        dhtp[iIdx] = 0;
        dvtp[iIdx] = 0;
    }
}

__global__ void TvDenoiseGPUStepOne(Matrix A, float *dhvt, float *dhtp, float *dvtp)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // 列坐标
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // 行坐标

    if(x_id < A.width && y_id < A.height)
    {
        int iIdx = x_id * A.height + y_id; // 该线程对应矩阵索引位置

        // 求 S = opQ(dhvt) = opQ(-opQt(pn) - g ./ lambda)
        float dh = dhvt[((x_id + 1) % A.width) * A.height + y_id] - dhvt[iIdx];
        float dv = dhvt[x_id * A.height + (y_id + 1) % A.height] - dhvt[iIdx];

        // 求 R = (1 + tau * modulo(S))
        float R = sqrt(dh * dh + dv * dv);

        A.elements[iIdx] += R;
    }
}


void TvNormSqrt(float *A, float *C, int iRow, int iCol, int band)
{
    Matrix d_A = createMat(iRow, iCol, band); // 指向显存的矩阵

    size_t oneBandSize = d_A.size * sizeof(float); // 一个谱段占据内存空间大小
    size_t allBandSize = d_A.band * oneBandSize; // 所有谱段占据内存空间大小

    // 分配Pageable(交换页)内存
    hipMalloc((void**)&d_A.elements, oneBandSize);// 在显存中为d_A开辟空间

    float *CHost, *AHost;
    // 分配Pinned(Page-locked)内存
    hipHostAlloc((void**)&CHost, oneBandSize, hipHostMallocDefault);
    hipHostAlloc((void**)&AHost, allBandSize, hipHostMallocDefault);

    // 把未锁页的内存A拷贝到锁页的内存AHost
    memcpy(AHost, A, allBandSize);

    // 核函数尺度
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((d_A.width + BLOCK_SIZE -1) / dimBlock.x,
                 (d_A.height + BLOCK_SIZE -1) / dimBlock.y);

    // 创建流
    hipStream_t stream; //流的结构体
    hipStreamCreate(&stream);

    for(int iBand = 0; iBand < d_A.band; iBand++)
    {
        int iBandOffset = iBand * d_A.size;

        hipMemcpyAsync(d_A.elements, AHost + iBandOffset, oneBandSize, hipMemcpyHostToDevice, stream);

        TvDenoiseGPUSetZero<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp); // 初始化htvt, dhtp, dvtp = 0;

        for (int iter = 0; iter < piter; ++iter)
        {
            TvDenoiseGPUStepOne<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp, lambda); // 求 dhvt = -opQt(pn) - g ./ lambda
            TvDenoiseGPUStepTwo<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp, lambda);
            hipStreamSynchronize(stream);
        }
        TvDenoiseGPUStepThree<<<dimGrid, dimBlock, 0, stream>>>(d_A, dhvt, dhtp, dvtp, lambda);
    }

        // 将d_A矩阵从显存中读到主机内存中
        hipMemcpyAsync(CHost, d_A.elements, oneBandSize, hipMemcpyDeviceToHost, stream);
    // 强制CUDA运行时等待流中所有操作完成
    hipStreamSynchronize(stream);

    // 把锁页的内存CHost拷贝到未锁页的内存C
    memcpy(C, CHost, oneBandSize);

    // 释放Pinned(Page-locked)内存
    hipHostFree(CHost);
    hipHostFree(AHost);

    // 释放显存空间
    hipFree(d_A.elements);
    hipFree(dhvt);
    hipFree(dhtp);
    hipFree(dvtp);

    hipDeviceReset();
}
