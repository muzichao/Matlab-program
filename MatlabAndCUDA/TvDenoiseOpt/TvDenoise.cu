#include "hip/hip_runtime.h"
#include "TvDenoise.h"
#include "D:\Matlab\extern\include\mex.h"

#define BLOCK_SIZE 16

// c��c++�еľ����������ȴ洢
// M.width ��ƫ��
// M.elements �����׵�ַ
// M(row, col) = *(M.elements + col * M.height + row)
typedef struct
			{
				int width;
				int height;
                size_t size;
				float* elements;
			} Matrix;

// ����������к������ɾ���ṹ��
Matrix createMat(const int height, const int width)
{
	Matrix mat;
	mat.height = height;
	mat.width = width;
	mat.size = height * width * sizeof(float);
	return mat;
}

__global__ void TvDenoiseGPU1(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda, int piter, float tau)
{
    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if(x_id < A.width && y_id < A.height)
	{
		int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

		// �� dhvt = opQt(pn)
		dhvt[iIdx] = dhtp[((x_id + A.width - 1) % A.width) * A.height + y_id] - dhtp[iIdx]
			         + dvtp[x_id * A.height + (y_id + A.height - 1) % A.height] - dvtp[iIdx];

		// �� dhvt = -opQt(pn) - g ./ lambda
		dhvt[iIdx] = - dhvt[iIdx] - A.elements[iIdx] / lambda;
	}
}

__global__ void TvDenoiseGPU2(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda, int piter, float tau)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if(x_id < A.width && y_id < A.height)
	{
		int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��

		// �� S = opQ(dhvt) = opQ(-opQt(pn) - g ./ lambda)
		float dh = dhvt[((x_id + 1) % A.width) * A.height + y_id] - dhvt[iIdx];
		float dv = dhvt[x_id * A.height + (y_id + 1) % A.height] - dhvt[iIdx];

		// �� R = (1 + tau * modulo(S))
		float R = 1 + tau * sqrt(dh * dh + dv * dv);

		// �� pn = [dht dvt]
		dhtp[iIdx] = (dhtp[iIdx] + tau * dh) / R;
		dvtp[iIdx] = (dvtp[iIdx] + tau * dv) / R;
	}
}

__global__ void TvDenoiseGPU3(Matrix A, float *dhvt, float *dhtp, float *dvtp, float lambda, int piter, float tau)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	if(x_id < A.width && y_id < A.height)
	{
		int iIdx = x_id * A.height + y_id; // ���̶߳�Ӧ��������λ��
		// �� dhvt = opQt(pn)
		dhvt[iIdx] = dhtp[((x_id + A.width - 1) % A.width) * A.height + y_id] - dhtp[iIdx]
		              + dvtp[x_id * A.height + (y_id + A.height - 1) % A.height] - dvtp[iIdx];
		// ��������д��A������
		A.elements[iIdx] += lambda * dhvt[iIdx];
	}
}

void TvDenoiseGPU(float *A, float *C, int iRow, int iCol, float lambda, int piter, float tau)
{
    Matrix d_A = createMat(iRow, iCol);
    hipMalloc((void**)&d_A.elements, d_A.size);// ���Դ���Ϊd_A���ٿռ�
    hipMemcpy(d_A.elements, A, d_A.size, hipMemcpyHostToDevice);

    float *dhvt; // �м����
    float *dhtp; // �м����
    float *dvtp; // �м����
	hipMalloc((void**)&dhvt, d_A.size);// ���Դ���Ϊdhvt���ٿռ�
    hipMalloc((void**)&dhtp, d_A.size);// ���Դ���Ϊdhtp���ٿռ�
    hipMalloc((void**)&dvtp, d_A.size);// ���Դ���Ϊdvtp���ٿռ�
	hipMemset(dhvt, 0, d_A.size); //��ʼ��dhvt
	hipMemset(dhtp, 0, d_A.size); //��ʼ��dhtp
	hipMemset(dvtp, 0, d_A.size); //��ʼ��dvtp

    // �˺���
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((d_A.width + BLOCK_SIZE -1) / dimBlock.x,
                 (d_A.height + BLOCK_SIZE -1) / dimBlock.y);

	for (int iter = 0; iter < piter; ++iter)
	{
		TvDenoiseGPU1<<<dimGrid, dimBlock>>>(d_A, dhvt, dhtp, dvtp, lambda, piter, tau); // �� dhvt = -opQt(pn) - g ./ lambda
		TvDenoiseGPU2<<<dimGrid, dimBlock>>>(d_A, dhvt, dhtp, dvtp, lambda, piter, tau);
	}
	TvDenoiseGPU3<<<dimGrid, dimBlock>>>(d_A, dhvt, dhtp, dvtp, lambda, piter, tau);

    // ��d_A������Դ��ж��������ڴ���
    hipMemcpy(C, d_A.elements, d_A.size, hipMemcpyDeviceToHost);

    // �ͷ��Դ�ռ�
    hipFree(d_A.elements);
	hipFree(dhvt);
	hipFree(dhtp);
	hipFree(dvtp);
}
