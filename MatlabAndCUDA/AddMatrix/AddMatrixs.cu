#include "hip/hip_runtime.h"
#include "AddMatrixs.h"
#include "D:\Matlab\extern\include\mex.h"

#define BLOCK_SIZE 32
__global__ void addMatrixsMask(float* A, float* B, float* C, int Row, int Col)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    
    if(i >= Row || j >= Col)
        return;

    //C[i][j] = A[i][j] + B[i][j];
    C[j * Row + i] = A[j * Row + i] + B[j * Row + i];
}

void addMatrixs(float* A, float* B, float* C, int Row, int Col)
{
    float *devPtrA = 0, *devPtrB = 0, *devPtrC = 0;
    
    hipMalloc(&devPtrA, sizeof(float) * Row * Col);
    hipMalloc(&devPtrB, sizeof(float) * Row * Col);
    hipMalloc(&devPtrC, sizeof(float) * Row * Col);
    
    hipMemcpy(devPtrA, A, sizeof(float) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(devPtrB, B, sizeof(float) * Row * Col, hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(int((Col + BLOCK_SIZE -1 )/BLOCK_SIZE),
                 int((Row + BLOCK_SIZE -1 )/BLOCK_SIZE));
    addMatrixsMask<<<dimGrid, dimBlock>>>(devPtrA, devPtrB, devPtrC, Row, Col);
    
    hipMemcpy(C, devPtrC, sizeof(float) * Row * Col, hipMemcpyDeviceToHost);
    
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);   
}
